#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <hipsparse.h>
#include <vector>
#include <fstream>
#include <nvtx3/nvToolsExt.h>
#include "common.h"

__global__ void warmup_kernel(int N, int* a)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = a[i];
  }
}

void print_ints(int N, int* a, const char* nvtx_name)
{
  nvtxRangePush(nvtx_name);
  for(int i = 0; i < N; ++i) {
    printf("element %d = %d\n", i, a[i]);
  }
  nvtxRangePop();
}

// CUDA example with kernel
void hello_world_test(const int N)
{
  printf("\n\n***Running %s\n\n", __func__);
  nvtxRangePush(__func__);

  int* a_m;
  CHECK_CUDA(hipMallocManaged(&a_m, sizeof(int)*N));
  for(int i = 0; i < N; ++i) {
    a_m[i] = -1;
  }

  printf("Printing managed memory in a_m pre-kernel\n");
  print_ints(N, a_m, "pre_copy_print");

  // Overwrite a_m with GPU to 1337
  // a_m = 1337
  // TODO Implement kernel

  printf("\nPrinting managed memory post-kernel\n");
  print_ints(N, a_m, "post_copy_print");

  CHECK_CUDA(hipFree(a_m));
  nvtxRangePop();
}

int main(int argc, char** argv)
{
  CHECK_CUDA(hipFree(0)); // Initialize the GPU context
  CHECK_CUDA(hipSetDevice(0)); // Choose device 0 to execute on

  int N = 1024*1024*1024;
  int* a;
  CHECK_CUDA(hipMalloc(&a, sizeof(int)*N));
  warmup_kernel<<<N/128, 128>>>(N, a);
  CHECK_CUDA(hipFree(a));
  CHECK_CUDA(hipDeviceSynchronize());
                                
  N = 10;

  hello_world_test(N);
}

