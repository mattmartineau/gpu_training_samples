#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <hipsparse.h>
#include <vector>
#include <fstream>
#include <nvtx3/nvToolsExt.h>
#include "common.h"

__global__ void warmup_kernel(int N, int* a)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = a[i];
  }
}

__global__ void initialize_a_red(int N, double* a, double val)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = val;
  }
}

__global__ void multipass_reduction_kernel(int N, double* a, double* sum_private)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i >= N) return;

  // TODO Implement
}

// CUDA example of multipass reduction
void reduction_multipass_first_test()
{
  printf("\n\n***Running %s\n\n", __func__);
  nvtxRangePush(__func__);

  // Large problem this time!
  int N = 1024*1024*1024;

  // Make space for the array we are reducing
  // Initialise all of the array to 1 (ignore sloppy launch params!)
  double* a;
  CHECK_CUDA(hipMalloc(&a, sizeof(double)*N));
  initialize_a_red<<<N/128, 128>>>(N, a, 1.0);
  CHECK_CUDA(hipDeviceSynchronize());

  // We're fixing the block sizes so we have more than one element per thread
  int block_size = 256;
  int nblocks = 64*1024;
  int nthreads = block_size*nblocks;

  // Only need nthreads private space (hint: that's ALOT!)
  double* sum_private;
  CHECK_CUDA(hipMalloc(&sum_private, sizeof(double)*nthreads));

  // Perform the reduction
  multipass_reduction_kernel<<<nblocks, block_size>>>(int N, double* a, double* sum_private)

  // Let thrust reduce the private sums
  double sum = thrust::reduce(thrust::device,
          sum_private, sum_private + nthreads, 0, thrust::plus<double>());

  // Check we validate
  if(sum != (double)N) {
      printf("Failed validation sum=%.2f, n=%.2f\n", sum, (double)N);
  }
  else {
      printf("Successful validation sum=%.2f, n=%.2f\n", sum, (double)N);
  }

  CHECK_CUDA(hipFree(a));
  CHECK_CUDA(hipFree(sum_private));
  nvtxRangePop();
}

int main(int argc, char** argv)
{
  CHECK_CUDA(hipFree(0)); // Initialize the GPU context
  CHECK_CUDA(hipSetDevice(0)); // Choose device 0 to execute on

  int N = 1024*1024*1024;
  int* a;
  CHECK_CUDA(hipMalloc(&a, sizeof(int)*N));
  warmup_kernel<<<N/128, 128>>>(N, a);
  CHECK_CUDA(hipFree(a));
  CHECK_CUDA(hipDeviceSynchronize());

  N = 512;
  
  reduction_multipass_first_test();
}

