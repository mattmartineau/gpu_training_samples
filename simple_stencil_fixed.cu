#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <hipsparse.h>
#include <vector>
#include <fstream>
#include <nvtx3/nvToolsExt.h>
#include "common.h"

__global__ void warmup_kernel(int N, int* a)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = a[i];
  }
}

__global__ void initialize_a(int N, int* a)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = i;
  }
}

__global__ void stencil_kernel(int N, int* a_orig, int* a_out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i >= N) return;

  if(i > 0) {
    a_out[i-1] += a_orig[i];
  }
  if(i < N-1) {
    a_out[i+1] += a_orig[i];
  }
}

// CUDA example with stencil
void simple_stencil_fixed_test(const int N)
{
  printf("\n\n***Running %s\n\n", __func__);
  nvtxRangePush(__func__);

  int* a_out;
  int* a_ref;
  int* a_orig;
  CHECK_CUDA(hipMallocManaged(&a_out, sizeof(int)*N));
  CHECK_CUDA(hipMallocManaged(&a_ref, sizeof(int)*N));
  CHECK_CUDA(hipMallocManaged(&a_orig, sizeof(int)*N));

  int block_size = 128;
  int nblocks = N/block_size + 1;
  initialize_a<<<nblocks, block_size>>>(N, a_orig);

  // hipMemcpy is naturally synchronous...
  CHECK_CUDA(hipMemcpy(a_out, a_orig, sizeof(int)*N, hipMemcpyDefault));
  CHECK_CUDA(hipMemcpy(a_ref, a_orig, sizeof(int)*N, hipMemcpyDefault));

  // CPU stencil kernel
  for(int i = 0; i < N; ++i) {
      if(i > 0) {
          a_ref[i-1] += a_orig[i];
      }
      if(i < N-1) {
          a_ref[i+1] += a_orig[i];
      }
  }

  // TODO Fix the kernel
  stencil_kernel<<<nblocks, block_size>>>(N, a_orig, a_out);
  CHECK_CUDA(hipDeviceSynchronize());

  // Validation
  int failures = 0;
  for(int i = 0; i < N; ++i) {
      if(a_out[i] != a_ref[i]) {
          failures++;
          printf("a_out[%d]=%d != a_ref[%d]=%d !!\n", i, a_out[i], i, a_ref[i]);
      }
  }

  if(failures == 0) {
      printf("Kernel validates!!\n");
  }

  CHECK_CUDA(hipFree(a_out));
  CHECK_CUDA(hipFree(a_orig));
  CHECK_CUDA(hipFree(a_ref));
  nvtxRangePop();
}

int main(int argc, char** argv)
{
  CHECK_CUDA(hipFree(0)); // Initialize the GPU context
  CHECK_CUDA(hipSetDevice(0)); // Choose device 0 to execute on

  int N = 1024*1024*1024;
  int* a;
  CHECK_CUDA(hipMalloc(&a, sizeof(int)*N));
  warmup_kernel<<<N/128, 128>>>(N, a);
  CHECK_CUDA(hipFree(a));
  CHECK_CUDA(hipDeviceSynchronize());
                                
  N = 512;
  simple_stencil_fixed_test(N);
}

