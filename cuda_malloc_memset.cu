#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <hipsparse.h>
#include <vector>
#include <fstream>
#include <nvtx3/nvToolsExt.h>
#include "common.h"

__global__ void warmup_kernel(int N, int* a)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < N) {
    a[i] = a[i];
  }
}

void print_ints(int N, int* a, const char* nvtx_name)
{
  nvtxRangePush(nvtx_name);
  for(int i = 0; i < N; ++i) {
    printf("element %d = %d\n", i, a[i]);
  }
  nvtxRangePop();
}

// CUDA example with no kernel
void cuda_malloc_memset_test(const int N)
{
  printf("\n\n***Running %s\n\n", __func__);

  // NVTX range (instrumentation) accepting function name
  nvtxRangePush(__func__);

  // Allocate some host memory and initialize
  int* a_h = (int*)malloc(sizeof(int)*N);
  for(int i = 0; i < N; ++i) {
    a_h[i] = -1;
  }

  // Initialize some device memory
  int* a_d;
  // TODO Allocate some device memory of size N

  // Initialize a_d with GPU to 0
  // a_d = 0
  // TODO Set that device memory to 0

  printf("Printing host memory a_h pre-copy\n");
  print_ints(N, a_h, "pre_copy_print");

  // Copy device data to host memory
  // a_h = a_d
  // TODO Copy that memory from the device to the host

  printf("\nPrinting host memory a_h post-copy\n");
  print_ints(N, a_h, "post_copy_print");

  // Free allocations on host and device
  delete[] a_h;
  // TODO Deallocate the GPU memory

  // End NVTX region
  nvtxRangePop();
}

int main(int argc, char** argv)
{
  CHECK_CUDA(hipFree(0)); // Initialize the GPU context
  CHECK_CUDA(hipSetDevice(0)); // Choose device 0 to execute on

  int N = 1024*1024*1024;
  int* a;
  CHECK_CUDA(hipMalloc(&a, sizeof(int)*N));
  warmup_kernel<<<N/128, 128>>>(N, a);
  CHECK_CUDA(hipFree(a));
  CHECK_CUDA(hipDeviceSynchronize());
                                
  N = 10;

  cuda_malloc_memset_test(N);
}

